#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif

#define PI  3.14159256

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------

__device__ float2 compute_fft(float *real_image, float *imag_image, float *fft_real, float *fft_imag, int size) {
  float real_value = 0;
  float imag_value = 0;

  for(unsigned int n = 0; n < size; n++)
  {
    real_value += (real_image[n] * fft_real[n]) - (imag_image[n] * fft_imag[n]);
    imag_value += (imag_image[n] * fft_real[n]) + (real_image[n] * fft_imag[n]);
  }

  float2 result;
  result.x = real_value;
  result.y = imag_value;

  return result;
}

__global__ void cuda_fftx(float *real_image, float *imag_image, int size_x, int size_y)
{
  int x = blockIdx.x; // each row of the image is processed by a different thread block
  int y = threadIdx.x; // each column in the row is processed by a different thread within the block

  __shared__ float real_image_buf[SIZEX]; // these shared buffers help in reducing the memory latency. Instead of fetching 
  __shared__ float imag_image_buf[SIZEX]; // image pixel data from global memory, the data can now be fetched from shared memory.

  // Populate the buffers in shared memory
  real_image_buf[y] = real_image[x*size_x + y];
  imag_image_buf[y] = imag_image[x*size_x + y];

  // Compute and store the required the cos/sine values.
  float fft_real[SIZEY];
  float fft_imag[SIZEY];

  for(unsigned int n = 0; n < size_y; n++)
  {
    float term = -2 * PI * y * n / size_y;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  __syncthreads();

  float2 fft = compute_fft(real_image_buf, imag_image_buf, fft_real, fft_imag, size_y);
  float real_value = fft.x;
  float imag_value = fft.y;
  
  real_image[x*size_x + y] = real_value;
  imag_image[x*size_x + y] = imag_value;

  // Reclaim memory
  delete [] fft_real;
  delete [] fft_imag;
}

__global__ void cuda_ffty(float *real_image, float *imag_image, int size_x, int size_y)
{
  int y = blockIdx.x; // each column is processed by a different thread block.
  int x = threadIdx.x; // each row in the column is processed by a different thread within the thread block.

  // Populate the buffers in shared memory
  __shared__ float real_image_buf[SIZEX];
  __shared__ float imag_image_buf[SIZEX];

  real_image_buf[x] = real_image[x*size_x + y];
  imag_image_buf[x] = imag_image[x*size_x + y];

  // Compute and store the required the cos/sine values.
  float fft_real[SIZEX];
  float fft_imag[SIZEX];

  for(unsigned int n = 0; n < size_y; n++)
  {
    float term = -2 * PI * x * n / size_x;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  __syncthreads();

  float2 fft = compute_fft(real_image_buf, imag_image_buf, fft_real, fft_imag, size_y);
  float real_value = fft.x;
  float imag_value = fft.y;

  real_image[x*size_x + y] = real_value;
  imag_image[x*size_x + y] = imag_value;

  // Reclaim memory
  delete [] fft_real;
  delete [] fft_imag;
}

__global__ void cuda_filter(float *real_image, float *imag_image, int size_x, int size_y)
{
  int eightX = size_x/8;
  int eightY = size_y/8;
  int eight7Y = size_y - eightY;

  int x = blockIdx.x; // each row is processed by a different thread block
  int y = threadIdx.x; // each column of the row is processed by a different thread.

  if(!(x < eightX && y < eightY) &&
        !(x < eightX && y >= eight7Y) &&
        !(x >= eight7Y && y < eightY) &&
        !(x >= eight7Y && y >= eight7Y))
  {
    // Zero out these values
    real_image[y*size_x + x] = 0;
    imag_image[y*size_x + x] = 0;
  }
}

__global__ void cuda_ifftx(float *real_image, float *imag_image, int size_x, int size_y)
{
  int x = blockIdx.x;
  int y = threadIdx.x;

  // Populate the buffers in shared memory
  __shared__ float real_image_buf[SIZEX];
  __shared__ float imag_image_buf[SIZEX];

  real_image_buf[y] = real_image[x*size_x + y];
  imag_image_buf[y] = imag_image[x*size_x + y];

  // Compute and store the required the cos/sine values.
  float fft_real[SIZEY];
  float fft_imag[SIZEY];

  for(unsigned int n = 0; n < size_y; n++)
  {
    // Note that the negative sign goes away for the term here
    float term = 2 * PI * y * n / size_y;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  __syncthreads();

  float2 fft = compute_fft(real_image_buf, imag_image_buf, fft_real, fft_imag, size_y);
  float real_value = fft.x;
  float imag_value = fft.y;

  real_image[x*size_x + y] = real_value/size_y;
  imag_image[x*size_x + y] = imag_value/size_y;

  // Reclaim memory
  delete [] fft_real;
  delete [] fft_imag;
}

__global__ void cuda_iffty(float *real_image, float *imag_image, int size_x, int size_y)
{
  int y = blockIdx.x;
  int x = threadIdx.x;

  // Populate the buffers in shared memory
  __shared__ float real_image_buf[SIZEX];
  __shared__ float imag_image_buf[SIZEX];

  real_image_buf[x] = real_image[x*size_x + y];
  imag_image_buf[x] = imag_image[x*size_x + y];

  // Compute and store the required the cos/sine values.
  float fft_real[SIZEX];
  float fft_imag[SIZEX];

  for(unsigned int n = 0; n < size_y; n++)
  {
    // Note that the negative sign goes away for the term
    float term = 2 * PI * x * n / size_x;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  __syncthreads();

  float2 fft = compute_fft(real_image_buf, imag_image_buf, fft_real, fft_imag, size_y);
  float real_value = fft.x;
  float imag_value = fft.y;

  real_image[x*size_x + y] = real_value/size_x;
  imag_image[x*size_x + y] = imag_value/size_x;

  // Reclaim memory
  delete [] fft_real;
  delete [] fft_imag;
}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------


__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  //exampleKernel<<<1,128,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  int numBlocks = SIZEY;
  int numThreadsPerBlock = SIZEY;

  cuda_fftx<<<numBlocks,numThreadsPerBlock,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cuda_ffty<<<numBlocks,numThreadsPerBlock,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cuda_filter<<<numBlocks,numThreadsPerBlock,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cuda_ifftx<<<numBlocks,numThreadsPerBlock,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cuda_iffty<<<numBlocks,numThreadsPerBlock,0,filterStream>>>(device_real,device_imag,size_x,size_y);

  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

